#include "hip/hip_runtime.h"
// nvcc cusparse.cu  -lcusparse -lcublas -o  cusparse
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "support.h"
#include "hipblas.h"
#include <hipsparse.h>
#include "cusparse_cublas_funcs.cu"
#include <time.h>

double randReal(double low, double high) {
    double d;
    d = (double) rand() / ((double) RAND_MAX + 1);
    d = (low + d * (high - low));
    d = ((int)(d * 1000)) / 1000.0;
    return d;
}

int generateRandomSparseMatrix(int m, int n, double *A, double ratio) {
    int nnz = 0;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if(randReal(0, 1) < ratio) {
               A[i*n+j] = randReal(0, 100);
               nnz += 1;
            }
            else {
               A[i*n+j] = 0;
            }
        }
    }
    return nnz;
}

void generateRandomVector(int n, double *A) {
    for (int i = 0; i < n; i++) {
        A[i] = randReal(0, 100);
    }
}

void printMatrix(int m, int n, double *A) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f,", A[i*n+j]);
        }
        printf("\n");
    }
}

void printMatrix_d(int m, int n, double *A) {
    double *h_A = (double *) malloc(sizeof(double) * m * n);
    checkCudaErrors(hipMemcpy(h_A, A, sizeof(double)*n*m, hipMemcpyDeviceToHost));
    printMatrix(m, n, h_A);
    free(h_A);
}

void printDVec(int n, double *A) {
    for (int i = 0; i < n; i++) {
        printf("%f, ", A[i]);
    }
}

void printIVec(int n, int *A) {
    for (int i = 0; i < n; i++) {
        printf("%d, ", A[i]);
    }
}

void printDVec_d(int n, double *A) {
    double *h_A = (double *) malloc(sizeof(double) * n);
    checkCudaErrors(hipMemcpy(h_A, A, sizeof(double)*n, hipMemcpyDeviceToHost));
    printDVec(n, h_A);
    free(h_A);
}

void printIVec_d(int n, int *A) {
    int *h_A = (int *) malloc(sizeof(int) * n);
    checkCudaErrors(hipMemcpy(h_A, A, sizeof(int)*n, hipMemcpyDeviceToHost));
    printIVec(n, h_A);
    free(h_A);
}


/* CPU version */

void vecAdd_cpu(int n, double *A, double *B, double *C){
    for (int i = 0; i < n; ++i)
        C[i] = A[i] + B[i];
}

double innerProduct_cpu(int n, double *A, double *B){
    double product = 0.0;
    for (int i = 0; i < n; ++i)
        product += A[i] * B[i];
    return product;
}

void scalarMulVec_cpu(int n, double val, double *A, double *B)
{
    for (int i = 0; i < n; ++i)
        B[i] = A[i] * val;
}

void matMulMat_cpu(int n, int m, int t, double *A, double *B, double *C){
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < t; ++j) {
            C[i*t+j] = 0;
            for (int k = 0; k < m; ++k)
                C[i*t+j] += A[i*m+k] * B[k*t+j];
        }
}

// calculate vector A* matrix B and store it in vector C
void vecMulMat_cpu(int n, int m, double *A, double *B, double *C) {
    for (int i = 0; i < m; ++i) {
        C[i] = 0;
        for (int j = 0; j < n; ++j)
            C[i] += A[j] * B[j*m+i];
    }
}


// calculate matrix A* vector B and store it in vector C
void matMulVec_cpu(int n, int m, double *A, double *B, double *C){
    for (int i = 0; i < n; ++i)
    {
        C[i] = 0;
        for (int j = 0; j < m; ++j)
            C[i] += A[i*m + j] * B[j];
    }

}

// calculate sum of two matrices
void matAdd_cpu(int n, int m, double *A, double *B, double *C){
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            C[i*m+j] = A[i*m+j] + B[i*m+j];
}

void scalarMulMat_cpu(int m, int n, double val, double *A, double *B)
{
    for (int i = 0; i < m * n; ++i)
        B[i] = A[i] * val;
}



void verifyVecAdd(int n, double *A, double *B, double *C_gpu) {
    const float relativeTolerance = 1e-6;
    double C_cpu[n];

    clock_t tic = clock(); 
    vecAdd_cpu(n, A, B, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);

  for(int i = 0; i < n; i++) {
    double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
    if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
      printf("TEST FAILED\n");
      exit(0);
    }
  }
  printf("TEST PASSED\n");
}

void verifyInnerProduct(int n, double *A, double *B, double C_gpu) {
    const float relativeTolerance = 1e-6;
    clock_t tic = clock(); 
    double C_cpu = innerProduct_cpu(n, A, B);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);
    double relativeError = (C_cpu - C_gpu)/ C_cpu;
    if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
      printf("TEST FAILED\n");
      exit(0);
    }
  printf("TEST PASSED\n");
}

void verifyScaleMulVec(int n, double val, double *A, double* C_gpu) {
    const float relativeTolerance = 1e-6;
    double C_cpu[n];

    clock_t tic = clock(); 
    scalarMulVec_cpu(n, val, A, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);
  

  for(int i = 0; i < n; i++) {
    double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
    if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
      printf("TEST FAILED\n");
      exit(0);
    }
  }
  printf("TEST PASSED\n");
}


void verifyMatMulVec(int m, int n, double *A, double *B, double *C_gpu) {
    const float relativeTolerance = 1e-6;
    double C_cpu[m];

    clock_t tic = clock(); 
    matMulVec_cpu(m, n, A, B, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);

      for(int i = 0; i < m; i++) {
        double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
        if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
          printf("TEST FAILED\n");
          exit(0);
        }
      }
      printf("TEST PASSED\n");
}

void verifyVecMulMat(int m, int n, double *A, double *B, double *C_gpu) {
    // B[1, m] x A[m, n] = C[1 , n]
    const float relativeTolerance = 1e-6;
    double C_cpu[n];

    clock_t tic = clock(); 
    vecMulMat_cpu(m, n, B, A, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);

      for(int i = 0; i < n; i++) {
        double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
        if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
          printf("TEST FAILED\n");
          exit(0);
        }
      }
      printf("TEST PASSED\n");
}

void verifyMatAdd(int m, int n, double *A, double *B, double *C_gpu) {
    // B[1, m] x A[m, n] = C[1 , n]
    const float relativeTolerance = 1e-6;
    double C_cpu[m*n];

    clock_t tic = clock(); 
    matAdd_cpu(m, n, A, B, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);

      for(int i = 0; i < m*n; i++) {
        double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
        if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
          printf("TEST FAILED\n");
          exit(0);
        }
      }
      printf("TEST PASSED\n");
}

void verifyMatMulMat(int m, int n, int k, double *A, double *B, double *C_gpu) {
    // B[1, m] x A[m, n] = C[1 , n]
    const float relativeTolerance = 1e-6;
    double C_cpu[m*k];

    clock_t tic = clock(); 
    matMulMat_cpu(m, n, k, A, B, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);

    for(int i = 0; i < m*k; i++) {
        double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
        if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
          printf("TEST FAILED\n");
          exit(0);
         }
    }
    printf("TEST PASSED\n");
}


void verifyScalarMulMat(int m, int n, double val, double *A, double* C_gpu) {
    const float relativeTolerance = 1e-6;
    double C_cpu[m*n];

    clock_t tic = clock(); 
    scalarMulMat_cpu(m, n, val, A, C_cpu);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("CPU time %fs, ", time_used);
  

    for(int i = 0; i < m*n; i++) {
        double relativeError = (C_cpu[i] - C_gpu[i])/ C_cpu[i];
        if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
          printf("TEST FAILED\n");
          exit(0);
        }
    }
  printf("TEST PASSED\n");
}



void testVecAdd(int n) {
    printf("- TEST vecAdd\n");
    double* h_A_val = (double*) malloc( sizeof(double)*n );
    double* h_B_val = (double*) malloc( sizeof(double)*n );
    generateRandomVector(n, h_A_val);
    generateRandomVector(n, h_B_val);

    // init vec with value
    VEC_h *h_A = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_A, n, h_A_val);

    VEC_h *h_B = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_B, n, h_B_val);

    // move to gpu vec_d
    VEC_d *d_A = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_A, d_A);

    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_B, d_B);

    //call function
    VEC_d *d_C = (VEC_d *) malloc(sizeof(VEC_d)); //no initialization

    clock_t tic = clock(); 
    vecAdd(d_A, d_B, d_C);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    // move to cpu
    VEC_h *h_C = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_d2h(d_C, h_C);
    verifyVecAdd(n, h_A->h_val, h_B->h_val, h_C->h_val);

    destroyVEC_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_d(d_C);
    destroyVEC_h(h_A);
    destroyVEC_h(h_B);
    destroyVEC_h(h_C);
}

void testVecAddInPlace(int n) {
    printf("- TEST vecAddInPlace\n");
    double* h_A_val = (double*) malloc( sizeof(double)*n );
    double* h_B_val = (double*) malloc( sizeof(double)*n );
    generateRandomVector(n, h_A_val);
    generateRandomVector(n, h_B_val);

    // init vec with value
    VEC_h *h_A = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_A, n, h_A_val);

    VEC_h *h_B = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_B, n, h_B_val);

    VEC_h *h_old_A = (VEC_h *) malloc(sizeof(VEC_h));
    copyVEC_h(h_A, h_old_A);

    // move to gpu vec_d
    VEC_d *d_A = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_A, d_A);

    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_B, d_B);

    clock_t tic = clock(); 
    vecAddInPlace(d_A, d_B);
    clock_t toc = clock();

    VEC_h *h_new_A = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_d2h(d_A, h_new_A);

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    // move to cpu
    verifyVecAdd(n, h_old_A->h_val, h_B->h_val, h_new_A->h_val);

    destroyVEC_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_h(h_A);
    destroyVEC_h(h_B);
    destroyVEC_h(h_old_A);
}

void testInnerProduct(int n) {
    printf("- TEST innerProduct\n");
    double* h_A_val = (double*) malloc( sizeof(double)*n );
    double* h_B_val = (double*) malloc( sizeof(double)*n );
    generateRandomVector(n, h_A_val);
    generateRandomVector(n, h_B_val);

    // init vec with value
    VEC_h *h_A = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_A, n, h_A_val);

    VEC_h *h_B = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_B, n, h_B_val);

    // move to gpu vec_d
    VEC_d *d_A = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_A, d_A);

    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_B, d_B);

    //call function
    clock_t tic = clock(); 
    double C = innerProduct(d_A, d_B);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    verifyInnerProduct(n, h_A->h_val, h_B->h_val, C);

    destroyVEC_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_h(h_A);
    destroyVEC_h(h_B);
}

void testScalarMulVec(int n, double sc) {
    printf("- TEST scalarMulVec\n");
    double* h_A_val = (double*) malloc( sizeof(double)*n );
    double* h_B_val = (double*) malloc( sizeof(double)*n );
    generateRandomVector(n, h_A_val);
    generateRandomVector(n, h_B_val);

    // init vec with value
    VEC_h *h_A = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_A, n, h_A_val);

    // move to gpu vec_d
    VEC_d *d_A = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_A, d_A);

    //call function
    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));

    clock_t tic = clock(); 
    scalarMulVec(sc, d_A, d_B);
    clock_t toc = clock();

    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    VEC_h *h_B = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_d2h(d_B, h_B);

    verifyScaleMulVec(n, sc, h_A->h_val, h_B->h_val);
    destroyVEC_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_h(h_A);
    destroyVEC_h(h_B);
}

void testMatMulVec(int m, int n) {
    printf("- TEST matMulVec\n");

    double* h_A_val = (double *) malloc(sizeof(double)*n*m);
    double* h_B_val = (double *) malloc(sizeof(double)*n);

    generateRandomSparseMatrix(m, n, h_A_val, 0.2);
    generateRandomVector(n, h_B_val);
    
    // init csr and vec
    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    VEC_h *h_B = (VEC_h*) malloc(sizeof(VEC_h));
    initVEC_h(h_B, n, h_B_val);

    // move to gpu
    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);

    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_B, d_B);

    // call function
    VEC_d *d_C = (VEC_d*) malloc(sizeof(VEC_d));

    clock_t tic = clock(); 
    matMulVec(d_A, d_B, d_C);
    clock_t toc = clock();
    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    VEC_h *h_C = (VEC_h*) malloc(sizeof(VEC_h));
    VEC_d2h(d_C, h_C);

    verifyMatMulVec(m, n, h_A_val, h_B->h_val, h_C->h_val);

    destroyCSR_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_d(d_C);
    destroyCSR_h(h_A);
    destroyVEC_h(h_B);
    destroyVEC_h(h_C);
    destroyDN_h(h_A_dn);
}

void testVecMulMat(int m, int n) {
    printf("- TEST matMulVec\n");

    double* h_A_val = (double *) malloc(sizeof(double)*n*m);
    double* h_B_val = (double *) malloc(sizeof(double)*m);

    generateRandomSparseMatrix(m, n, h_A_val, 0.2);
    generateRandomVector(m, h_B_val);
    
    // init csr and vec
    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    VEC_h *h_B = (VEC_h*) malloc(sizeof(VEC_h));
    initVEC_h(h_B, m, h_B_val);

    // move to gpu
    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);

    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_B, d_B);

    // call function
    VEC_d *d_C = (VEC_d*) malloc(sizeof(VEC_d));

    clock_t tic = clock(); 
    vecMulMat(d_B, d_A, d_C);
    clock_t toc = clock();
    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    VEC_h *h_C = (VEC_h*) malloc(sizeof(VEC_h));
    VEC_d2h(d_C, h_C);

    verifyVecMulMat(m, n, h_A_val, h_B->h_val, h_C->h_val);

    destroyCSR_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_d(d_C);
    destroyCSR_h(h_A);
    destroyVEC_h(h_B);
    destroyVEC_h(h_C);
    destroyDN_h(h_A_dn);
}

void testMatAdd(int m, int n) {
    printf("- TEST matAdd\n");

    double* h_A_val = (double *) malloc(sizeof(double)*n*m);
    double* h_B_val = (double *) malloc(sizeof(double)*n*m);

    generateRandomSparseMatrix(m, n, h_A_val, 0.2);
    generateRandomSparseMatrix(m, n, h_B_val, 0.2);
    
    // init csr and vec
    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    DN_h *h_B_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_B_dn, m, n, h_B_val);

    CSR_h *h_B = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_B_dn, h_B);

    // move to gpu
    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);
    CSR_d *d_B = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_B, d_B);

    // call function   
    CSR_d *d_C = (CSR_d *) malloc(sizeof(CSR_d));

    clock_t tic = clock(); 
    matAdd(d_B, d_A, d_C);
    clock_t toc = clock();
    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    CSR_h *h_C = (CSR_h*) malloc(sizeof(CSR_h));
    CSR_d2h(d_C, h_C);

    DN_h *h_C_dn = (DN_h *) malloc(sizeof(DN_h));
    CSR_h2DN_h(h_C, h_C_dn);

    verifyMatAdd(m, n, h_A_dn->h_val, h_B_dn->h_val, h_C_dn->h_val);

    destroyCSR_d(d_A);
    destroyCSR_d(d_B);
    destroyCSR_d(d_C);
    destroyCSR_h(h_A);
    destroyCSR_h(h_B);
    destroyCSR_h(h_C);
    destroyDN_h(h_A_dn);
    destroyDN_h(h_B_dn);
    destroyDN_h(h_C_dn);
}

void testScalarMulMat(int m, int n, double sc) {
    printf("- TEST matAdd\n");

    double* h_A_val = (double *) malloc(sizeof(double)*n*m);

    generateRandomSparseMatrix(m, n, h_A_val, 0.2);
    
    // init csr 
    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    // move to gpu
    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);

    // call function   
    CSR_d *d_C = (CSR_d *) malloc(sizeof(CSR_d));

    clock_t tic = clock(); 
    scalarMulMat(sc, d_A, d_C);
    clock_t toc = clock();
    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    CSR_h *h_C = (CSR_h*) malloc(sizeof(CSR_h));
    CSR_d2h(d_C, h_C);

    DN_h *h_C_dn = (DN_h *) malloc(sizeof(DN_h));
    CSR_h2DN_h(h_C, h_C_dn);

    verifyScalarMulMat(m, n, sc, h_A_dn->h_val, h_C_dn->h_val);

    destroyCSR_d(d_A);
    destroyCSR_d(d_C);
    destroyCSR_h(h_A);
    destroyCSR_h(h_C);
    destroyDN_h(h_A_dn);
    destroyDN_h(h_C_dn);
}

void testMatMulMat(int m, int n, int k) {
    printf("- TEST matMulMat\n");

    double* h_A_val = (double *) malloc(sizeof(double)*n*m);
    double* h_B_val = (double *) malloc(sizeof(double)*n*k);

    generateRandomSparseMatrix(m, n, h_A_val, 0.2);
    generateRandomSparseMatrix(n, k, h_B_val, 0.2);
    
    // init csr and vec
    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    DN_h *h_B_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_B_dn, n, k, h_B_val);

    CSR_h *h_B = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_B_dn, h_B);

    // move to gpu
    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);
    CSR_d *d_B = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_B, d_B);

    // call function   
    CSR_d *d_C = (CSR_d *) malloc(sizeof(CSR_d));

    clock_t tic = clock(); 
    matMulMat(d_A, d_B, d_C);
    clock_t toc = clock();
    double time_used = (double) (toc - tic) / CLOCKS_PER_SEC;
    printf("GPU time %fs, ", time_used);

    CSR_h *h_C = (CSR_h*) malloc(sizeof(CSR_h));
    CSR_d2h(d_C, h_C);

    DN_h *h_C_dn = (DN_h *) malloc(sizeof(DN_h));
    CSR_h2DN_h(h_C, h_C_dn);

    verifyMatMulMat(m, n, k, h_A_dn->h_val, h_B_dn->h_val, h_C_dn->h_val);

    destroyCSR_d(d_A);
    destroyCSR_d(d_B);
    destroyCSR_d(d_C);
    destroyCSR_h(h_A);
    destroyCSR_h(h_B);
    destroyCSR_h(h_C);
    destroyDN_h(h_A_dn);
    destroyDN_h(h_C_dn);
}

void testCopyVec(int n) {
    printf("- TEST copyVec\n");
    double* h_A_val = (double*) malloc( sizeof(double)*n );
    generateRandomVector(n, h_A_val);

    // init vec with value
    VEC_h *h_A = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_A, n, h_A_val);

    // move to gpu vec_d
    VEC_d *d_A = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_A, d_A);

    VEC_h *h_A_copy = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_d *d_A_copy = (VEC_d *) malloc(sizeof(VEC_d));

    copyVEC_h(h_A, h_A_copy);
    copyVEC_d(d_A, d_A_copy);
    
    printf("TEST PASSED\n");

    destroyVEC_d(d_A);
    destroyVEC_d(d_A_copy);
    destroyVEC_h(h_A);
    destroyVEC_h(h_A_copy);
}

void testCopyMat(int m, int n) {
    printf("- TEST copyMat\n");
    double* h_A_val = (double *) malloc(sizeof(double)*n*m);
    generateRandomSparseMatrix(m, n, h_A_val, 0.2);

    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);

    DN_h *h_A_dn_copy = (DN_h *) malloc(sizeof(DN_h));
    CSR_h *h_A_copy = (CSR_h *) malloc(sizeof(CSR_h));
    CSR_d *d_A_copy = (CSR_d *) malloc(sizeof(CSR_d));

    copyCSR_h(h_A, h_A_copy);
    copyCSR_d(d_A, d_A_copy);
    copyDN_h(h_A_dn, h_A_dn_copy);

    printf("TEST PASSED\n");
    
    destroyCSR_d(d_A);
    destroyCSR_d(d_A_copy);
    destroyCSR_h(h_A);
    destroyCSR_h(h_A_copy);
    destroyDN_h(h_A_dn);
    destroyDN_h(h_A_dn_copy);
}


void testVecUseCases() {
    double h_A_val[5] = {1, 2, 3, 4, 5};
    double h_B_val[5] = {2, 3, 4, 5, 6};
    double n = 5;

    // init vec with value
    VEC_h *h_A = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_A, n, h_A_val);

    VEC_h *h_B = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_B, n, h_B_val);

    // move to gpu vec_d
    VEC_d *d_A = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_A, d_A);

    VEC_d *d_B = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_B, d_B);

    printf("A:\n");
    printDVec(n, h_A_val);
    printf("\n");

    printf("B:\n");
    printDVec(n, h_B_val);
    printf("\n");

    // Add
    VEC_d *d_C_add = (VEC_d *) malloc(sizeof(VEC_d)); //no initialization 
    vecAdd(d_A, d_B, d_C_add);
    
    printf("Add:\n");
    printDVec_d(n, d_C_add->d_val);
    printf("\n");

    // Inner Product
    double c_inner = innerProduct(d_A, d_B);
    printf("Inner:\n");
    printf("%f\n", c_inner);

    // Scale
    VEC_d *d_C_sc = (VEC_d *) malloc(sizeof(VEC_d)); // no initialization 
    scalarMulVec(1.5, d_A, d_C_sc);
    printf("* 1.5:\n");
    printDVec_d(n, d_C_sc->d_val);
    printf("\n");

    // Add inplace
    vecAddInPlace(d_A, d_B);
    printf("Inplace Add:\n");
    printDVec_d(n, d_A->d_val);
    printf("\n");

    // Copy
    VEC_h *h_A_copy = (VEC_h *) malloc(sizeof(VEC_h));
    copyVEC_h(h_A, h_A_copy);
    printf("copy A_h:\n");
    printDVec(n, h_A_copy->h_val);
    printf("\n");

    VEC_d *d_A_copy = (VEC_d *) malloc(sizeof(VEC_d));
    copyVEC_d(d_A, d_A_copy);
    printf("copy A_d:\n");
    printDVec_d(n, d_A_copy->d_val);
    printf("\n");
    vecAddInPlace(d_A_copy, d_B);
    printDVec_d(n, d_A_copy->d_val);
    printf("\n");
    printDVec_d(n, d_A->d_val);
    printf("\n");

    destroyVEC_d(d_A);
    destroyVEC_d(d_B);
    destroyVEC_d(d_C_add);
    destroyVEC_d(d_A_copy);
}


void testMatUseCases() {
    double h_A_val[6] = {1, 2, 3, 4, 5, 6};
    double h_B_val[6] = {2, 3, 4, 5, 6, 7};

    double m = 3;
    double n = 2;
    
    // init csr and vec
    DN_h *h_A_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_A_dn, m, n, h_A_val);

    CSR_h *h_A = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_A_dn, h_A);

    DN_h *h_B_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_B_dn, m, n, h_B_val);

    CSR_h *h_B = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_B_dn, h_B);

    CSR_d *d_A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_A, d_A);

    CSR_d *d_B = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_B, d_B);

    printf("A:\n");
    printMatrix_d(m, n, d_A->d_val);
    printf("\n");

    printf("B:\n");
    printMatrix_d(m, n, d_B->d_val);
    printf("\n");

    printf("Add:\n");
    CSR_d *d_C_add = (CSR_d *) malloc(sizeof(CSR_d));
    matAdd(d_A, d_B, d_C_add);
    printMatrix_d(d_C_add->m, d_C_add->n, d_C_add->d_val);
    printf("\n");

    printf("A * 1.5:\n");
    CSR_d *d_C_sc = (CSR_d *) malloc(sizeof(CSR_d));
    scalarMulMat(1.5, d_A, d_C_sc);
    printMatrix_d(d_C_add->m, d_C_add->n, d_C_sc->d_val);
    printf("\n");

    double h_D_val[2] = {2, 3};
    VEC_h *h_D = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_D, n, h_D_val);
    VEC_d *d_D = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_D, d_D);
    printf("Vec D:\n");
    printDVec_d(d_D->n, d_D->d_val);
    printf("\n");
    VEC_d *d_C_mat_vec = (VEC_d *) malloc(sizeof(VEC_d));
    matMulVec(d_A, d_D, d_C_mat_vec);
    printf("Vec A * D:\n");
    printDVec_d(d_C_mat_vec->n, d_C_mat_vec->d_val);
    printf("\n");

    double h_E_val[3] = {2, 3, 4};
    VEC_h *h_E = (VEC_h *) malloc(sizeof(VEC_h));
    initVEC_h(h_E, m, h_E_val);
    VEC_d *d_E = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_h2d(h_E, d_E);
    printf("Vec E:\n");
    printDVec_d(d_E->n, d_E->d_val);
    printf("\n");
    VEC_d *d_C_vec_mat = (VEC_d *) malloc(sizeof(VEC_d));
    vecMulMat(d_E, d_A, d_C_vec_mat);
    printf("Vec A * E:\n");
    printDVec_d(d_C_vec_mat->n, d_C_vec_mat->d_val);
    printf("\n");

    double h_F_val[8] = {2, 3, 4, 5, 6, 7, 8, 9};
    double k = 4;
    DN_h *h_F_dn = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(h_F_dn, n, k, h_F_val);
    CSR_h *h_F = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(h_F_dn, h_F);
    CSR_d *d_F = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(h_F, d_F);

    printf("Mat F:\n");
    printMatrix_d(d_F->m, d_F->n, d_F->d_val);
    CSR_d *d_C_mat_mat = (CSR_d *) malloc(sizeof(CSR_d));
    matMulMat(d_A, d_F, d_C_mat_mat);
    printf("A * F\n");
    printMatrix_d(d_C_mat_mat->m, d_C_mat_mat->n, d_C_mat_mat->d_val);


}


int main(int argc, char**argv) {
    checkCublasErrors(hipblasCreate(&cublasHandle));
    checkCusparseErrors(hipsparseCreate(&cusparseHandle));

    testVecAdd(1000000);
    testVecAddInPlace(1000000);
    testInnerProduct(1000000);
    testScalarMulVec(1000000, 3.14159);
    testMatMulVec(1000, 500);
    testVecMulMat(1000, 500);
    testMatAdd(1000, 500);
    testMatMulMat(1000, 500, 100);
    testScalarMulMat(3.14159, 1000, 500);
    testCopyVec(1000000);
    testCopyMat(1000, 500);

    // testVecUseCases();
    // testMatUseCases();

    hipblasDestroy(cublasHandle);
    hipsparseDestroy(cusparseHandle);
 
    return 0;
}
 