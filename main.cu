#include "hip/hip_runtime.h"
//
// Created by Hantian Zhang on 4/4/2022.
//

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "cusparse_cublas_funcs.cu"

#define INF (1 << 28)
#define THREADS_PER_BLOCK   (1024)

# ifndef c_max
#  define c_max(a, b) (((a) > (b)) ? (a) : (b))
# endif /* ifndef c_max */

# ifndef c_min
#  define c_min(a, b) (((a) < (b)) ? (a) : (b))
# endif /* ifndef c_min */




__global__ void vec_bound_kernel(int n,
                              double *A_val,
                              double *l_val,
                              double *u_val,
                              double *B_val
){

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n) {
        B_val[i] = c_min(u_val[i], c_max(l_val[i], A_val[i]));
    }
}


void vecMinMaxProj(int n,
                   VEC_d *A,
                   VEC_d *l,
                   VEC_d *u,
                   VEC_d *B) {

    double *B_val;
    checkCudaErrors(hipMalloc((void**)&B_val, sizeof(double)*n));

    // printVecd(A);
    // printVecd(l);
    // printVecd(u);

    // printf("%d\n", n);
    int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;

    vec_bound_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(n, A->d_val, l->d_val, u->d_val, B_val);

    initVEC_d(B, n, B_val);

    // printVecd(A);
    // printVecd(l);
    // printVecd(u);
    // printVecd(B);
    // exit(0);
}

//// calculate transpose of a matrix A and store it in matrix B
void transpose(int n, int m, double *A, double *B){
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            B[j*n+i] = A[i*m+j];
}


void inverseDiag(CSR_h *A, CSR_h *B){
    copyCSR_h(A,B);
    for (int i =0; i < A->nnz; ++i)
    {
        B->h_val[i] = 1 / A->h_val[i];
    }
}



// fill a diagonal n*n matrix A with val on each diagonal entry
void initializeDiagMat(int n, double val, CSR_h *A){
    A->n = n;
    A->m = n;
    A->nnz = n;
    double *d_val = (double *) malloc(sizeof(double) *n);
    int *d_rowPtr = (int *) malloc(sizeof(int) *(n+1));
    int *d_colInd = (int *) malloc(sizeof(int) *n);
    for (int i = 0; i < n; ++i) {
        d_val[i] = val;
        d_rowPtr[i] = i;
        d_colInd[i] = i;
    }
    d_rowPtr[n] = n;
    A->h_rowPtr = d_rowPtr;
    A->h_colInd = d_colInd;
    A->h_val = d_val;
}

//// min(l) max(u) projection of an vector A to B
//void vecMinMaxProj(int n, double A[n], double l[n], double u[n], double B[n]){
//    for (int i = 0; i < n; ++i)
//        if (A[i] < l[i])
//            B[i] = l[i];
//        else if (A[i] > u[i])
//            B[i] = u[i];
//        else
//            B[i] = A[i];
//}


//double norm2(VEC_d *A){
//    double norm = 0.0;
//    double squared_sum = 0.0;
//    for (int i = 0; i < A->n; ++i)
//        squared_sum += A->d_val[i] * A->d_val[i];
//    norm = sqrt(squared_sum/A->n);
//    return norm;
//}


double normInf(VEC_d *A){
    int idx;
    double h_res;
    checkCublasErrors(hipblasIdamax(cublasHandle, A->n, A->d_val, 1, &idx));
    checkCudaErrors(hipMemcpy(&h_res, A->d_val + (idx-1), sizeof(double), hipMemcpyDeviceToHost));
    h_res = fabs(h_res);
    return h_res;
}


//double normInf(VEC_d *A){
//    double maxValue = -INF;
//    for (int i = 0; i < A->n; ++i)
//        if (A->d_val[i] > maxValue)
//            maxValue = A->d_val[i];
//    return maxValue;
//}


// if l==u, penalty = 1000*rho, otherwise penalty = rho
void calculateR(int n, double *R, double *l, double *u, double rho){
    memset(R, 0, sizeof(double)*n*n);
    for (int i = 0; i < n; ++i)
    {
        if (l[i] == u[i])
        {
            R[i*n+i] = rho * 1000;
        }
        else
        {
            R[i*n+i] = rho;
        }
    }
}

//get diagonal element of K into M
void getDiagonal(CSR_h *K, CSR_h *M)
{
    M->n = K->n;
    M->m = K->m;
    int nnz = 0;

    double *d_val = (double *) malloc(sizeof(double) *K->n);
    int *d_rowPtr = (int *) malloc(sizeof(int) * (K->n+1));
    int *d_colInd = (int *) malloc(sizeof(int) * (K->n));

    for (int i = 0; i < K->m; ++i) {
        int row_begin = K->h_rowPtr[i];
        int row_end = K->h_rowPtr[i+1];

        int count = 0;
        for (int j = row_begin; j < row_end; ++j) {
            if (K->h_colInd[j] == i) {
                d_val[nnz] = K->h_val[j];
                d_colInd[nnz] = i;
                nnz++;
                count++;
            }
        }
        if(i > 0) {
            d_rowPtr[i] = d_rowPtr[i-1]+count;
        }
        else {
            d_rowPtr[i] = 0;
        }
    }
    d_rowPtr[K->n] = nnz;

    M->h_rowPtr = d_rowPtr;
    M->h_colInd = d_colInd;
    M->h_val = d_val;
    M->nnz = nnz;
}

// calculate objective value obj = 1/2 * x^T * P * x + q^T * X
double objValue(int n, CSR_d *P, VEC_d *q, VEC_d *x)
{
    VEC_d *temp = (VEC_d *) malloc(sizeof(VEC_d));
    vecMulMat(x, P, temp);
    double obj = 0.5 * innerProduct(temp, x) + innerProduct(q, x);
    destroyVEC_d(temp);
    return obj;
}

// calculate if the program should terminate
int termination(VEC_d *x, VEC_d *y, VEC_d *z, CSR_d *P, VEC_d *Q, CSR_d *A, CSR_d *AT, double epsilonPrimal, double epsilonDual){
    // temp1 = A*x
    VEC_d *temp1 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp2 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *residualPrimal = (VEC_d *) malloc(sizeof(VEC_d));

    //calculate residual of Primal
    matMulVec(A, x, temp1); // temp1 = A*x;

    scalarMulVec(-1, z, temp2); // temp2 = -z;
    vecAdd(temp1, temp2, residualPrimal); // residualPrimal = A*x - z;

    //calculate residual of Dual
    VEC_d *temp3 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp4 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp5 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *residualDual = (VEC_d *) malloc(sizeof(VEC_d));

    matMulVec(P, x, temp3); // temp3 = P*x;
    matMulVec(AT, y, temp4); // temp4 = A^T * y;
    vecAdd(temp3, temp4, temp5); // temp5 = temp3 + temp4;
    vecAdd(temp5, Q, residualDual); // residualDual = temp5 + Q;
    double residualPrimalDouble = normInf(residualPrimal);
    double residualDualDouble = normInf(residualDual);
    destroyVEC_d(temp1);
    destroyVEC_d(temp2);
    destroyVEC_d(temp3);
    destroyVEC_d(temp4);
    destroyVEC_d(temp5);
    destroyVEC_d(residualPrimal);
    destroyVEC_d(residualDual);
    if ((residualPrimalDouble <= epsilonPrimal) && (residualDualDouble <= epsilonDual))
        return 1;
    else
        return 0;
}


// algorithm 3 with GPU
void solveKKT(int n, int m, VEC_d *x, VEC_d *y, VEC_d *z, CSR_d *P, VEC_d *Q, CSR_d *A, CSR_d *AT, VEC_d *l, VEC_d *u, CSR_d *R, VEC_d *xNext, VEC_d *zNext, double rho, double sigma, double epsilon)
{
    CSR_d *K = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_d *I = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h *I_h = (CSR_h *) malloc(sizeof(CSR_h));
    initializeDiagMat(n, 1, I_h);
    CSR_h2d(I_h, I);
    CSR_d *temp1 = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_d *temp2 = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_d *temp3 = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_d *temp4 = (CSR_d *) malloc(sizeof(CSR_d));
    matMulMat(AT, R, temp1); // temp1 = A^T * R;
    matMulMat(temp1, A, temp2); // temp2 = A^T * R * A;
    scalarMulMat(sigma, I, temp3);
    matAdd(temp2, temp3, temp4);
    matAdd(temp4, P, K);


    CSR_d *M = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_d *MINV = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h *M_h = (CSR_h *) malloc(sizeof(CSR_h));
    CSR_h *K_h = (CSR_h *) malloc(sizeof(CSR_h));

    CSR_d2h(K, K_h);
    getDiagonal(K_h, M_h);

    // printf("%d, %d, %d, %d, %d", MINV->m, MINV->n, r->n, MINV->nnz, M->nnz);
    // printDVec_d(M->nnz, M->d_val);
    // printDVec_d(MINV->nnz, MINV->d_val);

    CSR_h *MINV_h = (CSR_h *) malloc(sizeof(CSR_h));
    CSR_h2d(M_h, M);
    inverseDiag(M_h,MINV_h);
    CSR_h2d(MINV_h, MINV);

    VEC_d *b = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp5 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp6 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp7 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp8 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp9 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp10 = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *temp11 = (VEC_d *) malloc(sizeof(VEC_d));
    matMulVec(R, z, temp5);
    scalarMulVec(-1, y, temp6);
    vecAdd(temp5, temp6, temp7); // temp7 = Rz - y;
    matMulVec(AT, temp7, temp8); // temp8 = A^T(Rz - y);
    scalarMulVec(sigma, x, temp9); // temp9 = sigma * x;
    scalarMulVec(-1, Q, temp10); // temp10 = -q;
    vecAdd(temp9, temp10, temp11); // temp11 = sigma * x - q;
    vecAdd(temp11, temp8, b); // b = sigma * x - q + A^T(Rz - y);

    //initialize r0=Kx-b, y0, p0
    VEC_d *r = (VEC_d *) malloc(sizeof(VEC_d));
    matMulVec(K, x, temp8); // temp8 = Kx;
    scalarMulVec(-1, b, temp9); // temp9 = -b;

    // printCSRd(R);
    // printVecd(z);
    // printVecd(temp5);
    // printVecd(temp6);
    // printVecd(temp8);
    // printVecd(temp9);
    // printVecd(temp11);
    // printVecd(b);
    // exit(0);

    vecAdd(temp8, temp9, r);

    //intitialize y0 = M^(-1)*r;
    VEC_d *y_kkt = (VEC_d *) malloc(sizeof(VEC_d));

    matMulVec(MINV, r, y_kkt); // y0 = M^(-1)*r

    // intitialize p0 = -y0;
    VEC_d *p = (VEC_d *) malloc(sizeof(VEC_d));
    scalarMulVec(-1, y_kkt, p);

    // printf("r:");
    // printVecd(r);

    int k = 0;
    double normR = normInf(r);
    double normB = normInf(b);

    // printf("normR :%f\n", normR);
    // printf("normB :%f\n", normB);
    

    while (normR > epsilon * normB)
    {
        //calculate a^k
        double alpha;
        double temp12, temp13;
        temp12 = innerProduct(r, y_kkt); // calculate Numerator r^T * y;
        vecMulMat(p, K, temp8); // temp8 = p^T * K
        temp13 = innerProduct(temp8, p); // calculate denominator p^T * K * p;
        alpha = temp12 / temp13;
        //calculate x^k+1
        scalarMulVec(alpha, p, temp8);
   
        if (k==0) {
            vecAdd(x, temp8, xNext); // x = x + alpha * p;
        }
        else{
            vecAddInPlace(xNext, temp8);
        }
        //calculate r^k+1
        matMulVec(K, p, temp8); // temp8 = K * p;
        scalarMulVec(alpha, temp8, temp9); // temp9 = alpha * K * p;
        vecAddInPlace(r, temp9);

        //calculate y^k+1
        matMulVec(MINV, r, y_kkt); // y = M^(-1) * rNew;
        //calculate beta^k+1
        double beta;
        temp13 = innerProduct(r, y_kkt); // calculate Numerator rNew^T * yNew;
        beta = temp13 / temp12; // beta = ( rNew^T * yNew) / ( r^T * y)
        //calculate p^k+1
        scalarMulVec(-1, y_kkt, temp8); // temp8 = -y
        scalarMulVec(beta, p, temp9); // temp9 = beta * p;
        vecAdd(temp8, temp9, p);
        k+=1;

        normR = normInf(r);

//         printf("normR :%f\n", normR);
//         printf("normB :%f\n", normB);

    }
    matMulVec(A, xNext, zNext);

    // printVecd(xNext);
    // exit(0);

    destroyVEC_d(temp5);
    destroyVEC_d(temp6);
    destroyVEC_d(temp7);
    destroyVEC_d(temp8);
    destroyVEC_d(temp9);
    destroyVEC_d(temp10);
    destroyVEC_d(temp11);
    destroyVEC_d(b);
    destroyVEC_d(r);
    destroyVEC_d(y_kkt);
    destroyVEC_d(p);

    destroyCSR_d(temp1);
    destroyCSR_d(temp2);
    destroyCSR_d(temp3);
    destroyCSR_d(temp4);
    destroyCSR_d(K);
    destroyCSR_d(I);
    destroyCSR_d(M);
    destroyCSR_d(MINV);

    destroyCSR_h(M_h);
    destroyCSR_h(MINV_h);

    destroyCSR_h(K_h);
    destroyCSR_h(I_h);

}

//void twoD2oneD(int n, int m, double* A, double *B)
//{
//    for (int i = 0; i < n; i++)
//        for (int j = 0; j < m; j++)
//            B[i*n+j] = *((A+i*n) + j);
//}

//
void readSparseMatrix(char * FILENAME, CSR_h *DST, int *n, int *m)
{
    FILE *myfile;
    int nnz;
    myfile = fopen(FILENAME, "r");
    fscanf(myfile,"%i",n);
    fscanf(myfile,"%i",m);
    fscanf(myfile,"%i",&nnz);

    //printf("%d, %d, %d", *n, *m, nnz);
    double *h_val = (double *) malloc(sizeof(double) * nnz);
    int *h_rowPtr = (int *) malloc(sizeof(int) * (*n+1));
    int *h_colInd = (int *) malloc(sizeof(int) * nnz);
    for(int i = 0; i < nnz; i++)
    {
        fscanf(myfile,"%lf",h_val+i);
    }
//    for(int i = 0; i < nnz; i++)
//    {
//        printf("%lf,",h_val[i]);
//    }
//    printf("\n");
    for(int i = 0; i < *n + 1; i++)
    {
        fscanf(myfile,"%i",h_rowPtr+i);
    }
//    for(int i = 0; i < *n + 1; i++)
//    {
//        printf("%i,",h_rowPtr[i]);
//    }
//    printf("\n");
    for(int i = 0; i < nnz; i++)
    {
        fscanf(myfile,"%i",h_colInd+i);
    }
    initCSR_h(DST, *n, *m, nnz, h_val, h_rowPtr, h_colInd);
}

void readSparseMatrix(char * FILENAME, CSR_h *DST)
{
    FILE *myfile;
    int n, m, nnz;
    myfile = fopen(FILENAME, "r");
    fscanf(myfile,"%i",&n);
    fscanf(myfile,"%i",&m);
    fscanf(myfile,"%i",&nnz);

    //printf("%d, %d, %d\n", n, m, nnz);
    double *h_val = (double *) malloc(sizeof(double) * nnz);
    int *h_rowPtr = (int *) malloc(sizeof(int) * (n+1));
    int *h_colInd = (int *) malloc(sizeof(int) * nnz);
    for(int i = 0; i < nnz; i++)
    {
    fscanf(myfile,"%lf",h_val+i);
    }

    for(int i = 0; i < n + 1; i++)
    {
    fscanf(myfile,"%i",h_rowPtr+i);
    }
    for(int i = 0; i < nnz; i++)
    {
    fscanf(myfile,"%i",h_colInd+i);
    }
    initCSR_h(DST, n, m, nnz, h_val, h_rowPtr, h_colInd);

}

void readVector(char * FILENAME, VEC_h *DST)
{
    FILE *myfile;
    int n;
    myfile = fopen(FILENAME, "r");
    fscanf(myfile,"%i",&n);
    double *h_val = (double *) malloc(sizeof(double) * n);
    for(int i = 0; i < n; i++)
    {
        fscanf(myfile,"%lf",h_val+i);
    }
    initVEC_h(DST, n, h_val);
}

int main() {

    checkCublasErrors(hipblasCreate(&cublasHandle));
    checkCusparseErrors(hipsparseCreate(&cusparseHandle));
    //
    int n,m;
    double sigma = 0.000001, alpha=1.6, rho = 0.5;
    char testcase[30]="data/instance-300x400/";
    //get n and m
    // input
//    //case 1
//    n = 2;
//    m = 2;
//  case 2
//    n = 10;
//    m = 10;
//    double P_h[n*n],Q_h_val[n], A_h[m*n], AT_h[n*m], l_h_val[m],u_h_val[m];
    CSR_h *A_csrh = (CSR_h *) malloc(sizeof(CSR_h));
    CSR_h *AT_csrh = (CSR_h *) malloc(sizeof(CSR_h));
    CSR_h *P_csrh = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h *AT_dh_h = (DN_h *) malloc(sizeof(DN_h));
    DN_h *A_dh_h = (DN_h *) malloc(sizeof(DN_h));
    VEC_h *l_h = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_h *u_h = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_h *Q_h = (VEC_h *) malloc(sizeof(VEC_d));
    char testDST[30];
    strcpy(testDST, testcase);
    char filename[30] = "A.txt";
    strcat(testDST, filename);
    //printf("%s\n", testDST);
    readSparseMatrix(testDST,A_csrh,&m,&n);
    CSR_h2DN_h(A_csrh, A_dh_h);
    copyDN_h(A_dh_h, AT_dh_h);
    AT_dh_h->n = A_dh_h->m;
    AT_dh_h->m = A_dh_h->n;
    transpose(m, n, A_dh_h->h_val, AT_dh_h->h_val);
    DN_h2CSR_h(AT_dh_h, AT_csrh);

    strcpy(testDST, testcase);
    char filename_P[30] = "P.txt";
    strcat(testDST, filename_P);
    readSparseMatrix(testDST,P_csrh);

    strcpy(testDST, testcase);
    char filename_Q[30] = "q.txt";
    strcat(testDST, filename_Q);
    readVector(testDST,Q_h);

    strcpy(testDST, testcase);
    char filename_l[30] = "l.txt";
    strcat(testDST, filename_l);
    readVector(testDST,l_h);


    strcpy(testDST, testcase);
    char filename_u[30] = "u.txt";
    strcat(testDST, filename_u);
    readVector(testDST,u_h);
    //readSparseMatrix("")
    // case 1
//    P_h[0] = 0.01;
//    P_h[1] = 0.0;
//    P_h[2] = 0.0;
//    P_h[3] = 0.2889654;
//    Q_h_val[0] = -1.07296862;
//    Q_h_val[1] = 0.86540763;
//    A_h[0] = 0.0;
//    A_h[1] = 0.0;
//    A_h[2] = 0.0;
//    A_h[3] = -2.3015387;
//    AT_h[0] = 0.0;
//    AT_h[1] = 0.0;
//    AT_h[2] = 0.0;
//    AT_h[3] = -2.3015387;
//    l_h_val[0] = 0 - INF;
//    l_h_val[1] = 0 - INF;
//    u_h_val[0] = 0.22957721;
//    u_h_val[1] = -2.11756839;
 //   get_input(n,m,P,Q,l,u,A);
    //get AT = A^T
    //transpose(m,n, A_h, AT_h);
    //initialize x,y,z
    double R_h[m*m];
    calculateR(m, R_h, l_h->h_val, u_h->h_val, rho);
    //double A_2d[m*n];
    //double AT_2d[n*m];
    //double P_2d[n*n];

    //DN_h *A_dh_h = (DN_h *) malloc(sizeof(DN_h));
    //twoD2oneD(m, n, A_h, A_2d);
    //initDN_h(A_dh_h, m, n, A_h);
    //DN_h2CSR_h(A_dh_h, A_csrh);
    CSR_d *A = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(A_csrh, A);

    //DN_h *AT_dh_h = (DN_h *) malloc(sizeof(DN_h));
    //twoD2oneD(n, m, AT_h, AT_2d);
    //initDN_h(AT_dh_h, n, m, AT_h);
    //DN_h2CSR_h(AT_dh_h, AT_csrh);
    CSR_d *AT = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(AT_csrh, AT);

    //DN_h *P_dh_h = (DN_h *) malloc(sizeof(DN_h));
    //initDN_h(P_dh_h, n, n, P_h);
    //DN_h2CSR_h(P_dh_h, P_csrh);
    CSR_d *P = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(P_csrh, P);

    DN_h *R_dh_h = (DN_h *) malloc(sizeof(DN_h));
    initDN_h(R_dh_h, m, m, R_h);
    CSR_h *R_csrh = (CSR_h *) malloc(sizeof(CSR_h));
    DN_h2CSR_h(R_dh_h, R_csrh);
    CSR_d *R = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(R_csrh, R);

    CSR_h *RINV_h = (CSR_h *) malloc(sizeof(CSR_h));
    inverseDiag(R_csrh, RINV_h);
    CSR_d *RINV = (CSR_d *) malloc(sizeof(CSR_d));
    CSR_h2d(RINV_h, RINV);

    double x_h_val[n]; 
    double y_h_val[m]; 
    double z_h_val[m]; 
    memset(x_h_val, 0, sizeof(double)*n);
    memset(y_h_val, 0, sizeof(double)*m);
    memset(z_h_val, 0, sizeof(double)*m);

    VEC_h *x_h = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_h *y_h = (VEC_h *) malloc(sizeof(VEC_h));
    VEC_h *z_h = (VEC_h *) malloc(sizeof(VEC_h));


    initVEC_h(x_h, n, x_h_val);
    initVEC_h(y_h, m, y_h_val);
    initVEC_h(z_h, m, z_h_val);

    VEC_d *x = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *y = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *z = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *l = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *u = (VEC_d *) malloc(sizeof(VEC_d));
    VEC_d *Q = (VEC_d *) malloc(sizeof(VEC_d));

    VEC_h2d(x_h, x);
    VEC_h2d(y_h, y);
    VEC_h2d(z_h, z);
    VEC_h2d(l_h, l);
    VEC_h2d(u_h, u);
    VEC_h2d(Q_h, Q);

    // initialize sigma and alpha
    // initialize epsilon, now we assume epsilon is a constant
    // todo add a function to calculate epsilon in each iteration;
    double epsilon = 0.00001;
    double epsilonPrimal = 0.0001;
    double epsilonDual = 0.0001;
    //eps = calc_eps();
    int k = 0;

    while (!termination(x, y, z, P, Q, A, AT, epsilonPrimal, epsilonDual) && k <= 100)
    {

        VEC_d *xNext = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *zNext = (VEC_d *) malloc(sizeof(VEC_d));

        //calculate the penalty matrix R and its inverse RINV

        solveKKT(n, m, x, y, z, P, Q, A, AT, l,u, R, xNext, zNext, rho, sigma, epsilon);

        // update x
        VEC_d *temp1 = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *temp2 = (VEC_d *) malloc(sizeof(VEC_d));
//        printf("%d\n", k);
//        printVecd(xNext);
//        printVecd(zNext);
        scalarMulVec(alpha, xNext, temp1); // temp1 = alpha * xNext;
        scalarMulVec(1 - alpha, x, temp2); // temp2 = (1 - alpha) * x;
        vecAdd(temp1, temp2, x); // x = alpha * xNext +  (1 - alpha) * x;

        // update z
        VEC_d *temp3 = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *temp4 = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *temp5 = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *temp6 = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *temp7 = (VEC_d *) malloc(sizeof(VEC_d));
        VEC_d *zNextReal = (VEC_d *) malloc(sizeof(VEC_d));

        scalarMulVec(alpha, zNext, temp3); // temp3 = alpha * zNext;
        scalarMulVec(1 - alpha, z, temp4); // temp4 = (1 - alpha) * z;

        matMulVec(RINV,y, temp5 ); // temp5 = R^(-1) * y;

        
        vecAdd(temp3, temp4, temp6);  // temp6 = alpha * zNext + (1 - alpha) * z;
        vecAdd(temp5, temp6, temp7);    // temp7 = alpha * zNext + (1 - alpha) * z + R^(-1) * y

        // printVecd(xNext);
        // printVecd(zNext);
        // printVecd(temp1);
        // printVecd(temp2);
        // printVecd(temp3);
        // printVecd(temp4);
        // printVecd(temp5);
        // printVecd(temp6);
        // printVecd(temp7);
        // exit(0);

        vecMinMaxProj(l->n, temp7, l, u, zNextReal); // zNextReal is the projection of temp7

        // update y
        scalarMulVec(-1, zNextReal, temp3); // temp3 = -zNextReal;
        vecAdd(temp6, temp3, temp4); // temp4 = alpha * zNext + (1 - alpha) * z - zNextReal;
        matMulVec(R, temp4, temp5); // temp5 = R * (alpha * zNext + (1 - alpha) * z - zNextReal);
        vecAddInPlace(y, temp5); // y = y + temp5;
        
        // update z^k to z^(k+1)

        
        // printf("%d\n", zNextReal->n);
        // printVecd(zNextReal);
        copyVEC_d(zNextReal, z);

        printf("Round:%d, obj:%.6f\n", k, objValue(n,P,Q,x));
        //printVecd(x);
        k += 1;
   
        destroyVEC_d(temp1);
        destroyVEC_d(temp2);
        destroyVEC_d(temp3);
        destroyVEC_d(temp4);
        destroyVEC_d(temp5);
        destroyVEC_d(temp6);
        destroyVEC_d(temp7);
        destroyVEC_d(xNext);
        destroyVEC_d(zNext);
    }

    destroyVEC_d(x);
    destroyVEC_d(y);
    destroyVEC_d(z);
    destroyVEC_d(l);
    destroyVEC_d(u);
    destroyVEC_d(Q);
    destroyCSR_d(A);
    destroyCSR_d(AT);
    destroyCSR_d(R);
    destroyCSR_d(RINV);

    hipblasDestroy(cublasHandle);
    hipsparseDestroy(cusparseHandle);
 
    return 0;
}
